// -*- c++ -*-
//


#include <hip/hip_runtime.h>
#include <iostream>

// function to compute the invariants of a 3x3 tensor
__global__ void
computeInvariants(const double * A, double * I1, double * I2, double * I3, int size)
{
    // get the index of each thread
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    // make sure that the thread fits in the dimension
    if (index < size) {
        double A00 = A[index];
        double A01 = A[index + size];
        double A02 = A[index + 2 * size];
        double A10 = A[index + 3 * size];
        double A11 = A[index + 4 * size];
        double A12 = A[index + 5 * size];
        double A20 = A[index + 6 * size];
        double A21 = A[index + 7 * size];
        double A22 = A[index + 8 * size];

        // compute the first invariant
        I1[index] = A00 + A11 + A22;

        // compute the second invariant
        I2[index] = A00 * A11 + A11 * A22 + A00 * A22 - A01 * A10 - A02 * A20 - A12 * A21;

        // compute the third invariant
        I3[index] = A00 * (A11 * A22 - A12 * A21) - A01 * (A10 * A22 - A12 * A20)
                  + A02 * (A10 * A21 - A11 * A20);
    }

    return;
}

void
computeInvariantsManaged(
    int nTensors, int nThreadPerBlock, int nBlocks, const double * tensorArray, double * I1,
    double * I2, double * I3)
{
    // execute the kernel
    computeInvariants<<<nBlocks, nThreadPerBlock>>>(tensorArray, I1, I2, I3, nTensors);

    // all done
    return;
}

void
computeInvariantsPinned(
    int nTensors, int nThreadPerBlock, int nBlocks, const double * tensorArray, double * I1,
    double * I2, double * I3, double * gpuTensors, double * gpuI1, double * gpuI2, double * gpuI3)
{
    // set cuda error
    hipError_t status;

    // copy the pinned memory
    status =
        hipMemcpy(gpuTensors, tensorArray, nTensors * 9 * sizeof(double), hipMemcpyHostToDevice);
    if (status != hipSuccess) {
        // complain
        std::cout << "while sending memory " << hipGetErrorName(status) << " (" << status << ")"
                  << std::endl;
    }

    // execute the kernel
    computeInvariants<<<nBlocks, nThreadPerBlock>>>(gpuTensors, gpuI1, gpuI2, gpuI3, nTensors);

    // send the memory back
    status = hipMemcpy(I1, gpuI1, nTensors * sizeof(double), hipMemcpyDeviceToHost);
    if (status != hipSuccess) {
        // complain
        std::cout << "while receive memory " << hipGetErrorName(status) << " (" << status << ")"
                  << std::endl;
    }

    status = hipMemcpy(I2, gpuI2, nTensors * sizeof(double), hipMemcpyDeviceToHost);
    if (status != hipSuccess) {
        // complain
        std::cout << "while receive memory " << hipGetErrorName(status) << " (" << status << ")"
                  << std::endl;
    }

    status = hipMemcpy(I3, gpuI3, nTensors * sizeof(double), hipMemcpyDeviceToHost);
    if (status != hipSuccess) {
        // complain
        std::cout << "while receive memory " << hipGetErrorName(status) << " (" << status << ")"
                  << std::endl;
    }

    // all done
    return;
}

void
computeInvariantsMapped(
    int nTensors, int nThreadPerBlock, int nBlocks, const double * tensorArray, double * I1,
    double * I2, double * I3)
{
    // execute the kernel
    computeInvariants<<<nBlocks, nThreadPerBlock>>>(tensorArray, I1, I2, I3, nTensors);

    // all done
    return;
}

// end of file