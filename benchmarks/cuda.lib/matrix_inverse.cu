// -*- c++ -*-
//


#include <hip/hip_runtime.h>
#include <iostream>

// function to compute the invariants of a 3x3 tensor
__global__ void
computeInverse(const double * A, double * Ainv, int size)
{
    // get the index of each thread
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    // make sure that the thread fits in the dimension
    if (index < size) {
        double A00 = A[index];
        double A01 = A[index + size];
        double A02 = A[index + 2 * size];
        double A10 = A[index + 3 * size];
        double A11 = A[index + 4 * size];
        double A12 = A[index + 5 * size];
        double A20 = A[index + 6 * size];
        double A21 = A[index + 7 * size];
        double A22 = A[index + 8 * size];

        // compute the determinant of A
        double detA = A00 * (A11 * A22 - A12 * A21) - A01 * (A10 * A22 - A12 * A20)
                    + A02 * (A10 * A21 - A11 * A20);

        // set to its inverse
        double detInv = 1.0 / detA;

        // and return the inverse tensor
        Ainv[index] = detInv * (A11 * A22 - A12 * A21);
        Ainv[index + size] = detInv * (-A01 * A22 + A02 * A21);
        Ainv[index + 2 * size] = detInv * (A01 * A12 - A02 * A11);
        Ainv[index + 3 * size] = detInv * (-A10 * A22 + A12 * A20);
        Ainv[index + 4 * size] = detInv * (A00 * A22 - A02 * A20);
        Ainv[index + 5 * size] = detInv * (-A00 * A12 + A02 * A10);
        Ainv[index + 6 * size] = detInv * (A10 * A21 - A11 * A20);
        Ainv[index + 7 * size] = detInv * (-A00 * A21 + A01 * A20);
        Ainv[index + 8 * size] = detInv * (A00 * A11 - A01 * A10);
    }

    return;
}

void
wrapperManaged(
    int nTensors, int nThreadPerBlock, int nBlocks, const double * tensorArray, double * inverseArray)
{
    // execute the kernel
    computeInverse<<<nBlocks, nThreadPerBlock>>>(tensorArray, inverseArray, nTensors);

    // all done
    return;
}


void
wrapperPinned(
    int nTensors, int nThreadPerBlock, int nBlocks, const double * tensorArray, double * inverseArray,
    double * gpuTensors, double * gpuInverses)
{
    // set cuda error
    hipError_t status;


    // copy the pinned memory
    status =
        hipMemcpy(gpuTensors, tensorArray, nTensors * 9 * sizeof(double), hipMemcpyHostToDevice);
    if (status != hipSuccess) {
        // complain
        std::cout << "while sending memory " << hipGetErrorName(status) << " (" << status << ")"
                  << std::endl;
    }

    // execute the kernel
    computeInverse<<<nBlocks, nThreadPerBlock>>>(gpuTensors, gpuInverses, nTensors);

    // send the memory back
    status = hipMemcpy(
        inverseArray, gpuInverses, nTensors * 9 * sizeof(double), hipMemcpyDeviceToHost);
    if (status != hipSuccess) {
        // complain
        std::cout << "while receive memory " << hipGetErrorName(status) << " (" << status << ")"
                  << std::endl;
    }

    // all done
    return;
}

void
wrapperMapped(
    int nTensors, int nThreadPerBlock, int nBlocks, const double * tensorArray, double * inverseArray)
{
    // execute the kernel
    computeInverse<<<nBlocks, nThreadPerBlock>>>(tensorArray, inverseArray, nTensors);

    // all done
    return;
}

// end of file