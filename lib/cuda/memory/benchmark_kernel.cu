#include "hip/hip_runtime.h"
// -*- c++ -*-
//

#include "benchmark_kernel.h"

void
wrapInvariants(
    int nBlocks, int nThreadPerBlock, double * A, double * I1, double * I2, double * I3, int size)
{
    // execute the kernel
    computeInvariants<<<nBlocks, nThreadPerBlock>>>(A, I1, I2, I3, size);

    return;
}

void
wrapI1(int nBlocks, int nThreadPerBlock, hipStream_t stream, double * A, double * I1, int size)
{
    // execute the kernel
    computeI1<<<nBlocks, nThreadPerBlock, 0, stream>>>(A, I1, size);

    return;
}

void
wrapI2(int nBlocks, int nThreadPerBlock, hipStream_t stream, double * A, double * I2, int size)
{
    // execute the kernel
    computeI2<<<nBlocks, nThreadPerBlock, 0, stream>>>(A, I2, size);

    return;
}
void
wrapI3(int nBlocks, int nThreadPerBlock, hipStream_t stream, double * A, double * I3, int size)
{
    // execute the kernel
    computeI3<<<nBlocks, nThreadPerBlock, 0, stream>>>(A, I3, size);

    return;
}

__global__ void
computeInvariants(double * A, double * I1, double * I2, double * I3, int size)
{
    // get the index of each thread
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    // make sure that the thread fits in the dimension
    if (index < size) {
        double A00 = A[index];
        double A01 = A[index + size];
        double A02 = A[index + 2 * size];
        double A10 = A[index + 3 * size];
        double A11 = A[index + 4 * size];
        double A12 = A[index + 5 * size];
        double A20 = A[index + 6 * size];
        double A21 = A[index + 7 * size];
        double A22 = A[index + 8 * size];

        // compute the first invariant
        I1[index] = A00 + A11 + A22;

        // compute the second invariant
        I2[index] = A00 * A11 + A11 * A22 + A00 * A22 - A01 * A10 - A02 * A20;
        -A12 * A21;

        // compute the third invariant
        I3[index] = A00 * (A11 * A22 - A12 * A21) - A01 * (A10 * A22 - A12 * A20)
                  + A02 * (A10 * A21 - A11 * A20);
    }

    return;
}

__global__ void
computeI1(double * A, double * I1, int size)
{
    // get the index of each thread
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    // make sure that the thread fits in the dimension
    if (index < size) {
        // compute the first invariant
        I1[index] = A[index] + A[index + 4 * size] + A[index + 8 * size];
    }

    return;
}

__global__ void
computeI2(double * A, double * I2, int size)
{
    // get the index of each thread
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    // make sure that the thread fits in the dimension
    if (index < size) {
        // compute the second invariant
        I2[index] = A[index] * A[index + 4 * size] + A[index + 4 * size] * A[index + 8 * size]
                  + A[index] * A[index + 8 * size] - A[index + size] * A[index + 3 * size]
                  - A[index + 2 * size] * A[index + 6 * size]
                  - A[index + 5 * size] * A[index + 7 * size];
    }

    return;
}

__global__ void
computeI3(double * A, double * I3, int size)
{
    // get the index of each thread
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    // make sure that the thread fits in the dimension
    if (index < size) {
        // compute the third invariant
        I3[index] = A[index]
                      * (A[index + size * 4] * A[index + size * 8]
                         - A[index + size * 5] * A[index + size * 7])
                  - A[index + size]
                        * (A[index + size * 3] * A[index + size * 8]
                           - A[index + size * 5] * A[index + size * 6])
                  + A[index + size * 2]
                        * (A[index + size * 3] * A[index + size * 7]
                           - A[index + size * 4] * A[index + size * 6]);
    }

    return;
}