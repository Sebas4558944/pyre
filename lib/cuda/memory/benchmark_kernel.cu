#include "hip/hip_runtime.h"
// -*- coding: utf-8 -*-
//

#include "benchmark_kernel.h"

void
wrapInvariants(
    int nBlocks, int nThreadPerBlock, double * A, double * I1, double * I2, double * I3, int size)
{
    // Execute the kernel
    computeInvariants<<<nBlocks, nThreadPerBlock>>>(A, I1, I2, I3, size);

    return;
}

void
wrapI1(int nBlocks, int nThreadPerBlock, hipStream_t stream, double * A, double * I1, int size)
{
    // Execute the kernel
    computeI1<<<nBlocks, nThreadPerBlock, 0, stream>>>(A, I1, size);

    return;
}

void
wrapI2(int nBlocks, int nThreadPerBlock, hipStream_t stream, double * A, double * I2, int size)
{
    // Execute the kernel
    computeI2<<<nBlocks, nThreadPerBlock, 0, stream>>>(A, I2, size);

    return;
}
void
wrapI3(int nBlocks, int nThreadPerBlock, hipStream_t stream, double * A, double * I3, int size)
{
    // Execute the kernel
    computeI3<<<nBlocks, nThreadPerBlock, 0 stream>>>(A, I3, size);

    return;
}

__global__ void
computeInvariants(double * A, double * I1, double * I2, double * I3, int size)
{
    // Get the index of each thread
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    // Make sure that the thread fits in the dimension
    if (index < size) {
        // Compute the first invariant
        double I1[index] = A[index] + A[index + 4 * size] + A[index + 8 * size];

        // Compute the second invariant
        double I2[index] =
            A[index] * A[index + 4 * size] + A[index + 4 * size] * A[index + 8 * size]
            + A[index] * A[index + 8 * size] - A[index + size] * A[index + 3 * size]
            - A[index + 2 * size] * A[index + 6 * size] - A[index + 5 * size] * A[index + 7 * size];

        // Compute the third invariant
        double I3[index] = A[index]
                             * (A[index + size * 4] * A[index + size * 8]
                                - A[index + size * 5] * A[index + size * 7])
                         - A[index + size]
                               * (A[index + size * 3] * A[index + size * 8]
                                  - A[index + size * 5] * A[index + size * 6])
                         + A[index + size * 2]
                               * (A[index + size * 3] * A[index + size * 7]
                                  - A[index + size * 4] * A[index + size * 6]);
    }

    return;
}

__global__ void
computeI1(double * A, double * I1, int size)
{
    // Get the index of each thread
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    // Make sure that the thread fits in the dimension
    if (index < size) {
        // Compute the first invariant
        double I1[index] = A[index] + A[index + 4 * size] + A[index + 8 * size];
    }

    return;
}

__global__ void
computeI2(double * A, double * I2, int size)
{
    // Get the index of each thread
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    // Make sure that the thread fits in the dimension
    if (index < size) {
        // Compute the second invariant
        double I2[index] =
            A[index] * A[index + 4 * size] + A[index + 4 * size] * A[index + 8 * size]
            + A[index] * A[index + 8 * size] - A[index + size] * A[index + 3 * size]
            - A[index + 2 * size] * A[index + 6 * size] - A[index + 5 * size] * A[index + 7 * size];
    }

    return;
}

__global__ void
computeI3(double * A, double * I3, int size)
{
    // Get the index of each thread
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    // Make sure that the thread fits in the dimension
    if (index < size) {
        // Compute the third invariant
        double I3[index] = A[index]
                             * (A[index + size * 4] * A[index + size * 8]
                                - A[index + size * 5] * A[index + size * 7])
                         - A[index + size]
                               * (A[index + size * 3] * A[index + size * 8]
                                  - A[index + size * 5] * A[index + size * 6])
                         + A[index + size * 2]
                               * (A[index + size * 3] * A[index + size * 7]
                                  - A[index + size * 4] * A[index + size * 6]);
    }

    return;
}